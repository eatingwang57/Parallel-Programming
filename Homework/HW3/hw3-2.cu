
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>

#define B 32

const int INF = ((1 << 30) - 1);
void input(char *inFileName);
void output(char *outFileName);

void block_FW();
int ceil(int a, int b);

int v, e;
int padding;
int *D;

int main(int argc, char *argv[])
{
    input(argv[1]);
    printf("%d\n", padding);
    block_FW();
    output(argv[2]);
    return 0;
}

void input(char *fileName)
{
    FILE *file = fopen(fileName, "rb");
    fread(&v, sizeof(int), 1, file);
    fread(&e, sizeof(int), 1, file);
    printf("%d\n", v);
    padding = v + B - (v % B);
    D = (int *)malloc(sizeof(int) * padding * padding);

    // INIT
    for (int i = 0; i < padding; i++)
    {
        for (int j = 0; j < padding; j++)
        {
            if (i == j)
                D[i * padding + i] = 0;
            else
                D[i * padding + j] = INF;
        }
    }

    int edge[3];
    for (int i = 0; i < e; ++i)
    {
        fread(edge, sizeof(int), 3, file);
        D[edge[0] * padding + edge[1]] = edge[2];
    }
    fclose(file);
}

void output(char *fileName)
{
    FILE *outfile = fopen(fileName, "w");
    for (int i = 0; i < v; ++i)
    {
        for (int j = 0; j < v; ++j)
        {
            if (D[i * padding + j] >= INF)
                D[i * padding + j] = INF;
        }
        fwrite(&D[i * padding], sizeof(int), v, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void Phase1(int *Dist, int r, int v)
{
    int i = threadIdx.y + r * B;
    int j = threadIdx.x + r * B;

    if (i >= v || j >= v)
        return;

    __shared__ int sharedDist[B * B];
    int tIdxY = threadIdx.y * B;
    int idx = tIdxY + threadIdx.x;
    int idxD = i * v + j;
    sharedDist[idx] = Dist[idxD];
    __syncthreads();

    for (int k = 0; k < B; k++)
    {
        int idxY = tIdxY + k;
        int idxX = k * B + threadIdx.x;
        int candidate = sharedDist[idxY] + sharedDist[idxX];
        if (sharedDist[idx] > candidate)
        {
            sharedDist[idx] = candidate;
        }
        __syncthreads();
    }

    Dist[idxD] = sharedDist[idx];
    // __syncthreads();
}

__global__ void Phase2(int *Dist, int r, int v)
{
    //  對應到Phase1的那個pivot block
    if (blockIdx.y == r)
        return;

    int i = threadIdx.y + blockIdx.y * B;
    int j = threadIdx.x + blockIdx.y * B;
    int p_i = threadIdx.y + r * B;
    int p_j = threadIdx.x + r * B;

    // blockIdx.x = 0 for row, = 1 for column
    if (blockIdx.x == 0)
    {
        i = p_i;
    }
    else
    {
        j = p_j;
    }

    if (i >= v || j >= v)
        return;

    __shared__ int sharedPivot[B * B];
    __shared__ int sharedSelf[B * B];
    int tIdxY = threadIdx.y * B;
    int idx = tIdxY + threadIdx.x;
    int idxD = i * v + j;
    int idxPivot = p_i * v + p_j;
    sharedPivot[idx] = Dist[idxPivot];
    sharedSelf[idx] = Dist[idxD];
    __syncthreads();

    if (blockIdx.x == 0)
    {
        for (int k = 0; k < B; k++)
        {
            int idxY = tIdxY + k;
            int idxX = k * B + threadIdx.x;
            int candidate = sharedPivot[idxY] + sharedSelf[idxX];
            if (sharedSelf[idx] > candidate)
            {
                sharedSelf[idx] = candidate;
            }
        }
    }
    else
    {
        for (int k = 0; k < B; k++)
        {
            int idxY = tIdxY + k;
            int idxX = k * B + threadIdx.x;
            int candidate = sharedSelf[idxY] + sharedPivot[idxX];
            if (sharedSelf[idx] > candidate)
            {
                sharedSelf[idx] = candidate;
            }
        }
    }

    Dist[idxD] = sharedSelf[idx];
    // __syncthreads();
}

__global__ void Phase3(int *Dist, int r, int v)
{
    if ((blockIdx.x == r) || (blockIdx.y == r))
        return;

    int i = threadIdx.y + blockIdx.y * B;
    int j = threadIdx.x + blockIdx.x * B;

    if (i >= v || j >= v)
        return;

    __shared__ int sharedRow[B * B];
    __shared__ int sharedCol[B * B];
    int idxD = i * v + j;
    int pt = Dist[idxD];
    int tIdxY = threadIdx.y * B;
    int idx = tIdxY + threadIdx.x;
    int idxPivotI = i * v + (threadIdx.x + r * B);
    int idxPivotJ = (threadIdx.y + r * B) * v + j;

    sharedRow[idx] = Dist[idxPivotI];
    sharedCol[idx] = Dist[idxPivotJ];
    __syncthreads();

    for (int k = 0; k < B; k++)
    {
        int idxY = tIdxY + k;
        int idxX = k * B + threadIdx.x;
        int candidate = sharedRow[idxY] + sharedCol[idxX];
        if (pt > candidate)
        {
            pt = candidate;
        }
    }
    Dist[idxD] = pt;
}

void block_FW()
{
    int round = ceil(padding, B);
    int *g_D;
    size_t size = sizeof(int) * padding * padding;
    hipMalloc(&g_D, size);
    hipMemcpy(g_D, D, size, hipMemcpyHostToDevice);

    for (int r = 0; r < round; ++r)
    {
        // printf("%d %d\n", r, round);
        // fflush(stdout);

        dim3 block_num1(1, 1);
        dim3 thread_num(B, B);
        dim3 block_num2(2, round);
        dim3 block_num3(round, round);
        /* Phase 1*/
        Phase1<<<block_num1, thread_num>>>(g_D, r, padding);

        /* Phase 2*/
        Phase2<<<block_num2, thread_num>>>(g_D, r, padding);

        /* Phase 3*/
        Phase3<<<block_num3, thread_num>>>(g_D, r, padding);
    }
    hipMemcpy(D, g_D, padding * padding * sizeof(int), hipMemcpyDeviceToHost); //
}